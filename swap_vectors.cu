
#include <hip/hip_runtime.h>
#include <stdio.h>


const int DSIZE = 40960;
const int block_size = 256;
const int grid_size = DSIZE/block_size;


__global__ void vector_swap(float *A, float *B, int v_size) {

    // Express the vector index in terms of threads and blocks
    int idx =  threadIdx.x + blockDim.x * blockIdx.x;

    // Swap the vector elements - make sure you are not out of range
    float temp = 0;
    if (idx < v_size) {
        temp = A[idx];
        A[idx] = B[idx];
        B[idx] = temp;
    }
}


int main() {

    float *h_A, *h_B, *d_A, *d_B;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];

    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate memory for host and device pointers 
    hipMalloc(&d_A, DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*sizeof(float));

    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    printf("Pre Swap:\n");
    for (int i = 0; i < 5; i++) 
        printf("A[%d]: %f, B[%d]: %f\n", i, h_A[i], i, h_B[i]);
    vector_swap<<<grid_size, block_size>>>(d_A, d_B, DSIZE);

    // Copy back to host 
    hipMemcpy(h_A, d_A, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make sure swapping was successfull
    printf("Post Swap:\n");
    for (int i = 0; i < 5; i++) printf("A[%d]: %f, B[%d]: %f\n", i, h_A[i], i, h_B[i]);

    // Free the memory 
    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
